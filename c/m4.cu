
#include <hip/hip_runtime.h>
#include <stdio.h>

#define WIDTH 800
#define HEIGHT WIDTH

__global__ void kernel(uchar4 * pbo, double centerX, double centerY,
					   double zoom, unsigned maxIterations) {
	// Calculate the relative thread identifiers
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	// Calculate the pixel index for this thread
	unsigned index = y * WIDTH + x;
	// Find the top left C value based on the center
	double startX = centerX - WIDTH/2/zoom;
	double startY = centerY - HEIGHT/2/zoom;
	// Calculate the C value for this thread
	double cReal = startX + x/zoom;
	double cImaginary = startY + y/zoom;
	// Begin Mandelbrot
	// Much of this code is the same as openMP
	double realZ = cReal;
	double imaginaryZ = cImaginary;
	double imaginarySquared;
	double realSquared;
	if(index < WIDTH*HEIGHT) {  
		unsigned n;
		for(n=0; n<maxIterations; n++) {
			realSquared = realZ*realZ;
			imaginarySquared = imaginaryZ*imaginaryZ;
			// Determine distance of Z from the origin of the complex plane'
			//simplified from sqrt(realZ*realZ + imaginaryZ*imaginary) > 2
			if (realSquared + imaginarySquared > 4)
				break;
			// Calculate z= z^2 + C
			// See: http:warp.povusers.org/Mandelbrot/ for simplification of function
			imaginaryZ = 2*realZ*imaginaryZ + cImaginary;
			realZ = realSquared - imaginarySquared + cReal;
		}
		if (n == maxIterations)
			1;
			// sprintf(&pbo[index], "%c", 0);
		else
			2;
			// pbo[index] = 255 - (n * 10 % 255);
	}
}

// Chop up the screen into 8x8 pixel sections
// Spawn a thread for each pixel
extern const int blockWidth = 8;
extern const int blockHeight = 8;
extern const int numBlocksWidth = WIDTH/blockWidth;
extern const int numBlocksHeight = HEIGHT/blockHeight;
// Declare the cuda dimension parameters
extern dim3 blockSize(blockWidth, blockHeight);
extern dim3 numBlocks(numBlocksWidth, numBlocksHeight);
extern "C" void runCuda(uchar4* pos, unsigned maxIterations, double zoom,
						double startX, double startY) {
	// Allocate memory to store the color palette
	uchar4 * d_palette;
	hipMalloc(&d_palette, sizeof(uchar4)*(maxIterations+1));
	// cudaMemcpy(d_palette, palette, sizeof(uchar4)*(maxIterations+1), 
	// cudaMemcpyHostToDevice);
	// Call the kernel
	kernel<<<numBlocks,blockSize>>>(pos, startX, startY, zoom, maxIterations);
	// Deallocate the palette memory
	// cudaFree(d_palette);
}

int main(int argc, char *argv[]) {
	uchar4* pos = NULL;
	int itermax = atoi(argv[1]);
	runCuda(pos, itermax, 3, -1, 0);
}